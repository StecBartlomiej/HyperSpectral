#include "hip/hip_runtime.h"
#include "Image.hpp"

#include <cassert>
#include <Classification.hpp>
#include <EntityComponentSystem.hpp>
#include <filesystem>
#include <string>

#include <iostream>
#include <map>
#include <numeric>
#include <span>
#include <cmath>

#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

extern Coordinator coordinator;



[[nodiscard]] Entity CreateImage(const FilesystemPaths &paths)
{
    auto id = coordinator.CreateEntity();

    coordinator.AddComponent(id, paths);

    const auto opt_envi = LoadEnvi(paths.envi_header);
    if (!opt_envi.has_value())
    {
        const auto file = paths.envi_header.string();
        LOG_ERROR("CreateImage: failed to load ENVI file {}!", file);
        throw std::runtime_error{"Empty envi header"};
    }
    coordinator.AddComponent(id, opt_envi.value());
    coordinator.AddComponent(id, ImageSize{
        opt_envi->samples_per_image,
        opt_envi->lines_per_image,
        opt_envi->bands_number});


    LOG_INFO("Created image id={}", id);
    return id;
}

[[nodiscard]] std::shared_ptr<float[]> LoadImage(const std::filesystem::path &path, const EnviHeader &envi)
{
    std::ifstream file{path, std::ios_base::binary | std::ios::in};
    assert(file.is_open());
    return LoadImage(file, envi);
}

[[nodiscard]] std::shared_ptr<float[]> LoadImage(std::istream &iss, const EnviHeader &envi)
{
    switch (envi.data_type)
    {
        case DataType::BYTE:
            return LoadImageType<char>(iss, envi);
        case DataType::INT16:
            return LoadImageType<int16_t>(iss, envi);
        case DataType::INT32:
            return LoadImageType<int32_t>(iss, envi);
        case DataType::INT64:
            return LoadImageType<int64_t>(iss, envi);
        case DataType::UINT16:
            return LoadImageType<int16_t>(iss, envi);
        case DataType::UINT32:
            return LoadImageType<int32_t>(iss, envi);
        case DataType::UINT64:
            return LoadImageType<int64_t>(iss, envi);
        case DataType::FLOAT32:
            return LoadImageType<float>(iss, envi);
        case DataType::FLOAT64:
            return LoadImageType<double>(iss, envi);

        case DataType::COMPLEX32:
        case DataType::COMPLEX64:
        default:
            LOG_ERROR("LoadImage unsupported data type: {}", static_cast<int>(envi.data_type));
            return nullptr;
    }
    return nullptr;
}

CpuMatrix GetImageData(Entity entity)
{
    static std::map<Entity, std::weak_ptr<float[]>> loaded_img{};

    const auto &size = coordinator.GetComponent<ImageSize>(entity);

    const auto iter = loaded_img.find(entity);
    if (iter != loaded_img.end() && !iter->second.expired())
    {
        return CpuMatrix{size, iter->second.lock()};
    }

    const auto &path = coordinator.GetComponent<FilesystemPaths>(entity).img_data;
    const auto &envi = coordinator.GetComponent<EnviHeader>(entity);

    std::shared_ptr<float[]> ptr = LoadImage(path, envi);
    loaded_img[entity] = ptr;

    return CpuMatrix{size, std::move(ptr)};
}

__device__ float GetElement(const Matrix matrix, std::size_t y, std::size_t x)
{
    return matrix.data[y * matrix.pixels_width + x];
}

__device__ void SetElement(const Matrix matrix, std::size_t y, std::size_t x, float value)
{
    matrix.data[y * matrix.pixels_width + x] = value;
}

__device__ void AddElement(const Matrix matrix, std::size_t y, std::size_t x, float value)
{
    matrix.data[y * matrix.pixels_width + x] += value;
}

Matrix CpuMatrix::GetMatrix() const
{
    return Matrix{
        .bands_height = size.depth,
        .pixels_width = size.width * size.height,
        .data = data.get()
    };
}

__global__ void Mean(Matrix img, Matrix mean)
{
    const auto y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < img.bands_height)
    {
        float result = 0.0f;

        for (std::size_t x = 0; x < img.pixels_width; ++x)
        {
            result += GetElement(img, y, x);
        }
        result /= static_cast<float>(img.pixels_width);
        SetElement(mean, y, 0, result);
    }
}

__global__ void SumRows(Matrix img, Matrix sum)
{
    const auto y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < img.bands_height)
    {
        float result = 0.0f;

        for (std::size_t x = 0; x < img.pixels_width; ++x)
        {
            result += GetElement(img, y, x);
        }
        AddElement(sum, y, 0, result);
    }
}

__global__ void PieceWiseDivision(Matrix m, float divisor)
{
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < m.pixels_width && y < m.bands_height)
    {
        float result = GetElement(m, y, x) / divisor;
        SetElement(m, y, x, result);
    }
}

__global__ void SubtractMean(Matrix img, Matrix mean)
{
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < img.bands_height && x < img.pixels_width)
    {
        const float value = GetElement(img, y, x) - GetElement(mean, y, 0);
        SetElement(img, y, x, value);
    }
}


__global__ void MatMulTrans(const Matrix img, const Matrix result)
{
    const std::size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const std::size_t x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= result.pixels_width || y >= result.bands_height)
        return;

    float value = 0.f;
    for (std::size_t i = 0; i < img.pixels_width; ++i)
    {
        // X * X^T
        value += GetElement(img, y, i) * GetElement(img, x, i);
    }
    AddElement(result, y, x, value);
}

Matrix CovarianceMatrix(std::function<CpuMatrix(std::size_t)> LoadData, uint32_t max_height, uint32_t max_width, std::size_t data_count)
{
    // pixels_width = x = pixels_width = ImageSize.max_width * ImageSize.hegith, bands_height = y = bands_height= ImageSize.depth

    auto blocking_load_img = [&, max_height, max_width](std::size_t i, Matrix &img) -> ImageSize {
        auto [size, ptr] = LoadData(i);

        img.pixels_width = size.width * size.height;
        img.bands_height = size.depth;

        assert(img.bands_height <= max_height);
        assert(img.pixels_width <= max_width);

        CudaAssert(hipMemcpy(img.data, ptr.get(), size.height * size.width * size.depth * sizeof(float), hipMemcpyHostToDevice));
        return size;
    };

    Matrix img{0, 0, nullptr};
    Matrix mean{max_height, 1, nullptr};
    Matrix cov{max_height, max_height, nullptr};

    Matrix img_to_copy{0, 0, nullptr};

    CudaAssert(hipMalloc(&img.data, max_height * max_width * sizeof(float)));
    CudaAssert(hipMalloc(&img_to_copy.data, max_height * max_width * sizeof(float)));
    CudaAssert(hipMalloc(&mean.data, max_height * sizeof(float)));
    CudaAssert(hipMalloc(&cov.data, max_height * max_height * sizeof(float)));

    CudaAssert(hipMemset(mean.data, 0, max_height * sizeof(float)));
    CudaAssert(hipMemset(cov.data, 0, max_height * max_height * sizeof(float)));


    hipStream_t stream1;
    CudaAssert(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));


    dim3 threads_sum{1024};
    dim3 blocks_sum{(max_height / 1024) + 1};

    dim3 threads_division{1, 1024};
    dim3 blocks_division{1, (max_height / 1024) + 1};

    dim3 threads_division_2{32, 32};
    dim3 blocks_division_2{(max_height / 32) + 1, (max_height / 32) + 1};

    dim3 threads_subtract{64, 16};
    dim3 blocks_subtract{(max_height / 64) + 1, (max_width / 16) + 1};

    dim3 threads_matmul{64, 16};
    dim3 blocks_matmul{(max_height / 64) + 1, (max_width / 16) + 1};

    LOG_INFO("Start calculation of mean");

    blocking_load_img(0, img);
    for (std::size_t i = 0; i < data_count - 1; ++i)
    {
        SumRows<<<blocks_sum, threads_sum, 0, stream1>>>(img, mean);

        // Load in parallel
        blocking_load_img(i + 1, img_to_copy);

        hipStreamSynchronize(stream1);
        std::swap(img, img_to_copy);
    }
    SumRows<<<blocks_sum, threads_sum, 0, stream1>>>(img, mean);
    PieceWiseDivision<<<blocks_division, threads_division, 0, stream1>>>(mean, static_cast<float>(img.pixels_width * data_count));
    CudaAssert(hipStreamSynchronize(stream1));

    blocking_load_img(0, img);
    for (std::size_t i = 0; i < data_count - 1; ++i)
    {
        SubtractMean<<<blocks_subtract, threads_subtract, 0, stream1>>>(img, mean);
        MatMulTrans<<<blocks_matmul, threads_matmul, 0, stream1>>>(img, cov);

        // Load in parallel
        blocking_load_img(i + 1, img_to_copy);

        hipStreamSynchronize(stream1);
        std::swap(img, img_to_copy);
    }
    SubtractMean<<<blocks_subtract, threads_subtract, 0, stream1>>>(img, mean);
    MatMulTrans<<<blocks_matmul, threads_matmul, 0, stream1>>>(img, cov);
    CudaAssert(hipStreamSynchronize(stream1));

    PieceWiseDivision<<<blocks_division_2, threads_division_2, 0, stream1>>>(cov, static_cast<float>(img.pixels_width * data_count));
    CudaAssert(hipStreamSynchronize(stream1));
    LOG_INFO("End covariance matrix");

    CudaAssert(hipFree(img.data));
    CudaAssert(hipFree(mean.data));
    CudaAssert(hipFree(img_to_copy.data));

    CudaAssert(hipStreamDestroy(stream1));

    return cov;
}

ResultPCA PCA(std::function<CpuMatrix(std::size_t)> LoadData, uint32_t max_height, uint32_t max_width, std::size_t data_count)
{
    hipStream_t stream1;
    CudaAssert(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));

    LOG_INFO("Start covariance matrix calculation");
    Matrix cov = CovarianceMatrix(LoadData, max_height, max_width, data_count);
    LOG_INFO("Ended covariance matrix calculation");

    float *d_eigenvalues = nullptr;
    CudaAssert(hipMalloc(&d_eigenvalues, max_height * sizeof(float)));

    // Calculate eigenvalues
    hipsolverHandle_t handle = nullptr;
    int *dev_info = nullptr;
    int lwork = 0; // size of workspace
    float *d_work = nullptr;
    constexpr hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    constexpr hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    const int size = static_cast<int>(cov.bands_height);

    CusolverAssert(hipsolverDnCreate(&handle));
    CudaAssert(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
    CusolverAssert(hipsolverSetStream(handle, stream1));

    CudaAssert(hipMallocAsync(&dev_info, sizeof(int), stream1));

    CusolverAssert(
        hipsolverDnSsyevd_bufferSize(handle, jobz, uplo, size, cov.data, size, d_eigenvalues, &lwork) );
    CudaAssert(hipMalloc(&d_work, sizeof(float) * lwork));

    CusolverAssert(
        hipsolverDnSsyevd(handle, jobz, uplo, size, cov.data, size, d_eigenvalues, d_work, lwork, dev_info) );

    int info = 0;
    CudaAssert(hipMemcpyAsync(&info, dev_info, sizeof(int), hipMemcpyDeviceToHost, stream1));
    CudaAssert(hipStreamSynchronize(stream1));
    LOG_INFO("PCA: CusolverDnSsyevd info = {}", info);
    if (info < 0)
    {
        LOG_WARN("PCA: {}-th parameter is wrong", -info);
    }
    CudaAssert(hipFree(d_work));
    CudaAssert(hipFree(dev_info));


    auto eigenvector = std::make_shared<float[]>(cov.bands_height * cov.pixels_width);
    auto eigenvalues = std::make_shared<float[]>(cov.pixels_width);

    hipMemcpy(eigenvector.get(), cov.data, cov.bands_height * cov.pixels_width * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(eigenvalues.get(), d_eigenvalues, cov.pixels_width * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    CudaAssert(hipFree(d_eigenvalues));

    CusolverAssert(hipsolverDnDestroy(handle));
    CudaAssert(hipStreamDestroy(stream1));

    CudaAssert(hipDeviceReset());

    CpuMatrix mat_eigenvalues{
        .size = ImageSize{
            .width = 1,
            .height = static_cast<uint32_t>(cov.bands_height),
            .depth = 1},
        .data = std::move(eigenvalues)
    };
    CpuMatrix mat_eigenvectors{
        .size = ImageSize{
            .width = static_cast<uint32_t>(cov.pixels_width),
            .height = static_cast<uint32_t>(cov.bands_height),
            .depth = 1},
        .data = std::move(eigenvector)
    };


    return {.eigenvalues = mat_eigenvalues, .eigenvectors = mat_eigenvectors};
}

__global__ void Threshold(Matrix img, std::size_t band, float threshold, float *mask)
{
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < img.pixels_width)
    {
        const float value = GetElement(img, band, x) > threshold ? 1.f : 0.f;
        mask[x] = value;
    }
}

CpuMatrix ManualThresholding(Matrix img, std::size_t band, float threshold)
{
    Matrix d_img{img.bands_height, img.pixels_width, nullptr};
    float *d_mask = nullptr;

    CudaAssert(hipMalloc(&d_img.data, img.bands_height * img.pixels_width * sizeof(float)));
    CudaAssert(hipMalloc(&d_mask, img.pixels_width * sizeof(float)));

    CudaAssert(hipMemcpy(d_img.data, img.data, img.bands_height * img.pixels_width * sizeof(float), hipMemcpyHostToDevice));
    CudaAssert(hipMemset(d_mask, 0, img.pixels_width * sizeof(float)));

    dim3 threads_mean{1024};
    dim3 blocks_mean{static_cast<unsigned int>(img.pixels_width) / 1024 + 1};
    Threshold<<<blocks_mean, threads_mean>>>(d_img, band, threshold, d_mask);
    CudaAssert(hipDeviceSynchronize());

    std::shared_ptr<float[]> mask{new float[img.pixels_width]};

    CudaAssert(hipMemcpy(mask.get(), d_mask, img.pixels_width * sizeof(float), hipMemcpyDeviceToHost));

    CudaAssert(hipFree(d_img.data));
    CudaAssert(hipFree(d_mask));

    ImageSize img_size = {
        .width = static_cast<uint32_t>(img.pixels_width),
        .height = static_cast<uint32_t>(1),
        .depth = 1};

    return {img_size, std::move(mask)};
}

std::size_t SumAll(Matrix img)
{
    return static_cast<std::size_t>(std::accumulate(img.data, img.data + img.pixels_width + img.pixels_width * (img.bands_height - 1), 0.f));
}
__global__ void ConcatNeighboursBand(Matrix old_img, ImageSize old_size, Matrix new_img, ImageSize new_size)
{
    static constexpr std::size_t up_left_offset =     1;
    static constexpr std::size_t up_center_offset =   2;
    static constexpr std::size_t up_right_offset =    3;
    static constexpr std::size_t mid_left_offset =    4;
    static constexpr std::size_t mid_right_offset =   5;
    static constexpr std::size_t down_left_offset =   6;
    static constexpr std::size_t down_center_offset = 7;
    static constexpr std::size_t down_right_offset =  8;

    static constexpr int max_x_threads = 1024;
    static constexpr int block_height = 3;
    static constexpr int max_block_width = max_x_threads;

    const auto block_start = blockIdx.x * (blockDim.x - 2);
    const auto x = block_start + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;
    const auto band = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= old_size.width || y >= new_size.height || band >= old_img.bands_height)
        return;

    const auto old_i = (y + 1) * old_size.width + x;
    const auto old_up_i = old_i - old_size.width;
    const auto old_down_i = old_i + old_size.width;

    const int block_width = (block_start + max_x_threads < old_size.width) ? max_block_width : (old_size.width - block_start);

    __shared__ float temp[max_block_width * block_height];

    // Fill upper row
    temp[threadIdx.x] = GetElement(old_img, band, old_up_i);

    // Fill center
    temp[block_width + threadIdx.x] = GetElement(old_img, band, old_i);

    // Fill lower row
    temp[block_width * 2 + threadIdx.x] = GetElement(old_img, band, old_down_i);

    __syncthreads();

    if (threadIdx.x == 0 || threadIdx.x == block_width - 1)
        return;

    // for the second line in the same with
    const auto temp_i = threadIdx.x; // value in range [1, block_width-2]

    const auto up_left =   temp[temp_i - 1];
    const auto up_center = temp[temp_i];
    const auto up_right =  temp[temp_i + 1];

    const auto mid_left =   temp[block_width + temp_i - 1];
    const auto mid_center = temp[block_width + temp_i];
    const auto mid_right =  temp[block_width + temp_i + 1];

    const auto down_left =   temp[block_width * 2 + temp_i - 1];
    const auto down_center = temp[block_width * 2 + temp_i];
    const auto down_right =  temp[block_width * 2 + temp_i + 1];

    const int band_offset = old_img.bands_height;

    const auto i = y * new_size.width + x - 1;
    SetElement(new_img, band, i, mid_center);

    // Neighbours bands
    SetElement(new_img, band + band_offset * up_left_offset,   i, up_left);
    SetElement(new_img, band + band_offset * up_center_offset, i, up_center);
    SetElement(new_img, band + band_offset * up_right_offset,  i, up_right);

    SetElement(new_img, band + band_offset * mid_left_offset,  i, mid_left);
    SetElement(new_img, band + band_offset * mid_right_offset, i, mid_right);

    SetElement(new_img, band + band_offset * down_left_offset,   i, down_left);
    SetElement(new_img, band + band_offset * down_center_offset, i, down_center);
    SetElement(new_img, band + band_offset * down_right_offset,  i, down_right);
}

CpuMatrix AddNeighboursBand(Matrix img, ImageSize size)
{
    ImageSize new_size{
        .width = size.width - 2,
        .height = size.height - 2,
        .depth = size.depth * 9
    };

    Matrix old_img{img.bands_height, img.pixels_width, nullptr};
    Matrix new_img{new_size.depth, new_size.width * new_size.height, nullptr};

    CudaAssert(hipMalloc(&old_img.data, old_img.bands_height * old_img.pixels_width * sizeof(float)));
    CudaAssert(hipMalloc(&new_img.data, new_img.bands_height * new_img.pixels_width * sizeof(float)));

    CudaAssert(hipMemcpy(old_img.data, img.data, old_img.bands_height * old_img.pixels_width * sizeof(float), hipMemcpyHostToDevice));

    dim3 threads{1024, 1, 1};
    dim3 blocks{
        static_cast<unsigned int>(size.width / 1024 + 1),
        static_cast<unsigned int>(new_size.height),
        static_cast<unsigned int>(old_img.bands_height)
    };
    ConcatNeighboursBand<<<blocks, threads>>>(old_img, size, new_img, new_size);
    hipFree(old_img.data);

    CpuMatrix cpu_matrix{
        new_size,
        std::shared_ptr<float[]>(new float[new_size.width * new_size.height * new_size.depth])
    };

    CudaAssert(hipMemcpy(cpu_matrix.data.get(), new_img.data, sizeof(float) * new_img.bands_height * new_img.pixels_width, hipMemcpyDeviceToHost));

    hipFree(new_img.data);

    return std::move(cpu_matrix);
}

__global__ void MulImages(Matrix img, std::size_t* position, std::size_t pos_size, Matrix output)
{
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < pos_size && y < img.bands_height)
    {
        auto pixel_pos = position[x];
        auto value = GetElement(img, y, pixel_pos);
        SetElement(output, y, x, value);
    }
}

std::vector<std::size_t> PositionFromMask(Matrix mask)
{
    assert(mask.pixels_width > 0);
    assert(mask.bands_height == 1);

    std::vector<std::size_t> position;
    for (std::size_t i = 0; i < mask.pixels_width; ++i)
    {
        if (mask.data[i] != 0)
        {
            position.push_back(i);
        }
    }
    return position;
}

CpuMatrix GetObjectFromMask(Matrix img, Matrix mask)
{
    assert(img.pixels_width == mask.pixels_width);

    const std::vector<std::size_t> position = PositionFromMask(mask);
    const std::size_t pixels = position.size();

    Matrix new_img{img.bands_height, pixels, nullptr};
    std::size_t *m_pos = nullptr;
    Matrix old_img{img.bands_height, img.pixels_width, nullptr};

    CudaAssert(hipMalloc(&old_img.data, old_img.bands_height * old_img.pixels_width * sizeof(float)));
    CudaAssert(hipMalloc(&m_pos, pixels * sizeof(std::size_t)));
    CudaAssert(hipMalloc(&new_img.data, new_img.bands_height * new_img.pixels_width * sizeof(float)));

    CudaAssert(hipMemcpy(old_img.data, img.data, old_img.bands_height * old_img.pixels_width * sizeof(float), hipMemcpyHostToDevice));
    CudaAssert(hipMemcpy(m_pos, position.data(), pixels * sizeof(std::size_t), hipMemcpyHostToDevice));

    dim3 threads_mean{32, 32};
    dim3 blocks_mean{static_cast<unsigned int>(pixels) / 32 + 1, static_cast<unsigned int>(old_img.bands_height / 32 + 1)};
    MulImages<<<blocks_mean, threads_mean>>>(old_img, m_pos, pixels, new_img);

    std::shared_ptr<float[]> cpu_ptr = std::make_shared<float[]>(new_img.bands_height * pixels);

    CudaAssert(hipMemcpy(cpu_ptr.get(), new_img.data, new_img.bands_height * pixels * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(old_img.data);
    hipFree(m_pos);
    hipFree(new_img.data);

    ImageSize size{1, static_cast<uint32_t>(new_img.pixels_width), static_cast<uint32_t>(new_img.bands_height)};

    return {size, std::move(cpu_ptr)};
}

__global__ void MatMul(const Matrix a, const Matrix b, const Matrix c)
{
    assert(a.pixels_width == b.bands_height);
    assert(a.bands_height == c.bands_height);
    assert(b.pixels_width== c.pixels_width);

    const std::size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const std::size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= c.pixels_width || y >= c.bands_height)
        return;

    float value = 0.f;
    for (std::size_t i = 0; i < a.pixels_width; ++i)
    {
        value += GetElement(a, y, i) * GetElement(b, i, x);
    }
    SetElement(c, y, x, value);
}


std::vector<CpuMatrix> MatmulPcaEigenvectors(const CpuMatrix &eigenvectors, std::size_t k_bands,
    std::function<CpuMatrix(std::size_t)> LoadData, uint32_t max_pixels, std::size_t data_count)
{
    // Matmul [k_bands, bands] x [bands, pixels]

    const auto bands = eigenvectors.size.width;
    assert(data_count >= 1);
    assert(k_bands < bands);

    auto blocking_load_img = [&, max_pixels](std::size_t i, Matrix &img) -> ImageSize {
        auto [size, ptr] = LoadData(i);

        img.pixels_width = size.width * size.height;
        img.bands_height = size.depth;

        assert(img.pixels_width <= max_pixels);
        assert(img.bands_height == bands);

        CudaAssert(hipMemcpy(img.data, ptr.get(), size.height * size.width * size.depth * sizeof(float), hipMemcpyHostToDevice));
        return size;
    };

    auto GetCpuMatrix = [k_bands](Matrix img, ImageSize size) -> CpuMatrix {
        ImageSize cpu_size = {.width = size.width, .height = size.height, .depth = static_cast<uint32_t>(k_bands)};

        std::shared_ptr<float[]> cpu_ptr = std::make_shared<float[]>(k_bands * size.width * size.height);
        CudaAssert(hipMemcpy(cpu_ptr.get(), img.data, k_bands * size.width * size.height * sizeof(float), hipMemcpyDeviceToHost));

        return CpuMatrix{cpu_size, std::move(cpu_ptr)};
    };



    Matrix c_eigenvectors{k_bands, bands, nullptr};
    Matrix c_img{bands, max_pixels, nullptr};
    Matrix c_img_to_copy{bands, max_pixels, nullptr};
    Matrix c_result{k_bands, max_pixels, nullptr};

    CudaAssert(hipMalloc(&c_eigenvectors.data, k_bands * bands * sizeof(float)));
    CudaAssert(hipMalloc(&c_img.data, bands * max_pixels * sizeof(float)));
    CudaAssert(hipMalloc(&c_img_to_copy.data, bands * max_pixels * sizeof(float)));
    CudaAssert(hipMalloc(&c_result.data, k_bands * max_pixels * sizeof(float)));

    CudaAssert(hipMemcpy(c_eigenvectors.data, eigenvectors.data.get(), k_bands * bands * sizeof(float), hipMemcpyHostToDevice));
    CudaAssert(hipMemset(c_result.data, 0.f, k_bands * max_pixels * sizeof(float)));

    hipStream_t stream1;
    CudaAssert(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));

    dim3 threads{32, 32};
    dim3 blocks{static_cast<unsigned int>(max_pixels) / 32 + 1, static_cast<unsigned int>(k_bands / 32 + 1)};

    std::vector<CpuMatrix> results;

    ImageSize loaded_img_size = blocking_load_img(0, c_img);
    c_result.pixels_width = loaded_img_size.width * loaded_img_size.height;

    for (std::size_t i = 0; i < data_count - 1; ++i)
    {
        MatMul<<<blocks, threads, 0, stream1>>>(c_eigenvectors, c_img, c_result);

        ImageSize loaded_img_size2 = blocking_load_img(i + 1, c_img_to_copy);

        // wait for stream
        hipStreamSynchronize(stream1);
        results.push_back(GetCpuMatrix(c_result, loaded_img_size));

        std::swap(c_img, c_img_to_copy);
        std::swap(loaded_img_size, loaded_img_size2);
        c_result.pixels_width = loaded_img_size.width * loaded_img_size.height;
    }
    MatMul<<<blocks, threads, 0, stream1>>>(c_eigenvectors, c_img, c_result);
    hipStreamSynchronize(stream1);
    results.push_back(GetCpuMatrix(c_result, loaded_img_size));

    hipStreamDestroy(stream1);
    hipFree(c_eigenvectors.data);
    hipFree(c_img.data);
    hipFree(c_img_to_copy.data);
    hipFree(c_result.data);

    return results;
}


__global__ void CalculateFourMovements(Matrix img, Matrix result)
{
    const auto y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < img.bands_height)
    {
        float sum2 = 0;
        float sum3 = 0;
        float sum4 = 0;

        for (std::size_t x = 0; x < img.pixels_width; ++x)
        {
            const auto pixel = GetElement(img, y, x);

            const float val2 = pixel * pixel;
            const float val3 = val2 * pixel;
            const float val4 = val3 * pixel;

            sum2 += val2;
            sum3 += val3;
            sum4 += val4;
        }

        sum2 /= static_cast<float>(img.pixels_width);
        sum3 /= static_cast<float>(img.pixels_width);
        sum4 /= static_cast<float>(img.pixels_width);

        SetElement(result, y, 0, sum2);
        SetElement(result, y, 1, sum3);
        SetElement(result, y, 2, sum4);
    }
}

std::vector<StatisticalParameters> GetStatistics(const CpuMatrix& cpu_img)
{
    assert(cpu_img.data != nullptr);

    Matrix img{cpu_img.size.depth, cpu_img.size.width * cpu_img.size.height, nullptr};
    Matrix mean{img.bands_height, 1, nullptr};

    Matrix four_movements{img.bands_height, 3, nullptr};

    CudaAssert(hipMalloc(&img.data, img.bands_height * img.pixels_width * sizeof(float)));
    CudaAssert(hipMalloc(&mean.data, mean.bands_height * mean.pixels_width * sizeof(float)));
    CudaAssert(hipMalloc(&four_movements.data, four_movements.bands_height * four_movements.pixels_width * sizeof(float)));

    CudaAssert(hipMemcpy(img.data, cpu_img.data.get(), img.bands_height * img.pixels_width * sizeof(float), hipMemcpyHostToDevice));
    CudaAssert(hipMemset(mean.data, 0, mean.bands_height * mean.pixels_width * sizeof(float)));

    dim3 threads_sum{1024};
    dim3 blocks_sum{static_cast<unsigned int>(img.bands_height/ 32 + 1)};

    dim3 threads_division{1, 1024};
    dim3 blocks_division{1, static_cast<unsigned int>(mean.bands_height/ 32 + 1)};

    dim3 threads_subtract{32, 32};
    dim3 blocks_subtract{static_cast<unsigned int>(img.pixels_width / 32 + 1), static_cast<unsigned int>(img.bands_height / 32 + 1)};

    dim3 threads_movement{1024};
    dim3 blocks_movement{static_cast<unsigned int>(img.bands_height/ 32 + 1)};


    /// START CUDA PIPELINE
    SumRows<<<blocks_sum, threads_sum>>>(img, mean);

    PieceWiseDivision<<<blocks_division, threads_division>>>(mean, static_cast<float>(img.pixels_width));

    SubtractMean<<<blocks_subtract, threads_subtract>>>(img, mean);

    CalculateFourMovements<<<blocks_movement, threads_movement>>>(img, four_movements);

    hipDeviceSynchronize();
    /// END CUDA PIPELINE


    std::unique_ptr<float[]> cpu_mean{new float[img.bands_height]};
    std::unique_ptr<float[]> cpu_movements{new float[img.bands_height * 3]};

    CudaAssert(hipMemcpy(cpu_mean.get(), mean.data, img.bands_height *  sizeof(float), hipMemcpyDeviceToHost));
    CudaAssert(hipMemcpy(cpu_movements.get(), four_movements.data, img.bands_height * 3 * sizeof(float), hipMemcpyDeviceToHost));

    std::vector<StatisticalParameters> result;
    for (std::size_t i = 0; i < img.bands_height; ++i)
    {
        const std::size_t idx = i * 3;

        const float mean_value = cpu_mean[i];
        const float second_movement = cpu_movements[idx]; // variance
        const float third_movement = cpu_movements[idx + 1];
        const float fourth_movement = cpu_movements[idx + 2];

        const float std_dev = sqrt(second_movement);

        const float skewness = third_movement / std::pow(std_dev, 3);
        const float kurtosis = fourth_movement / std::pow(std_dev, 4);

        result.push_back(StatisticalParameters{mean_value, second_movement, skewness, kurtosis});
    }

    hipFree(img.data);
    hipFree(mean.data);
    hipFree(four_movements.data);

    return result;
}

ImageLabel::ImageLabel(const std::filesystem::path &file_path, const ImageSize size): img_size_{size}
{
    assert(!file_path.empty());
    assert(size.width > 0);
    assert(size.height > 0);

    std::ifstream file(file_path);
    if (!file.is_open())
    {
        LOG_ERROR("ImageLabel: Could not open file {}", file_path.string());
        throw std::runtime_error("Could not open file");
    }

    LOG_INFO("Loading {} labels", img_size_.width * img_size_.height);
    image_label_.resize(img_size_.width * img_size_.height);
    for (std::size_t i = 0; i < img_size_.width * img_size_.height; ++i)
    {
        file >> image_label_[i];
    }
}

uint8_t ImageLabel::GetLabels(const PatchData patch_pos)
{
    const auto [x, y] = patch_pos;
    const auto idx = y * img_size_.width + x;
    return image_label_.at(idx);
}

CpuMatrix GetImportantEigenvectors(const CpuMatrix &eigenvectors, std::size_t k_bands)
{
    const auto [width, height, band] = eigenvectors.size;

    const int ptr_diff = (static_cast<int>(height) - static_cast<int>(k_bands)) * static_cast<int>(width);

    std::shared_ptr<float[]> eigenvectors_data(new float[k_bands * width]);
    memcpy(eigenvectors_data.get(), eigenvectors.data.get() + ptr_diff, k_bands * width * sizeof(float));

    ImageSize size{width, static_cast<uint32_t>(k_bands), 1};
    return CpuMatrix{size, std::move(eigenvectors_data)};
}

float SumAllCuda(Matrix data)
{
    float *c_ptr = nullptr;
    CudaAssert(hipMalloc(&c_ptr, static_cast<size_t>(data.bands_height) * static_cast<size_t>(data.pixels_width) * sizeof(float)));
    CudaAssert(hipMemcpy(c_ptr, data.data, data.bands_height * data.pixels_width * sizeof(float), hipMemcpyHostToDevice));

    thrust::device_vector<float> c_vec(c_ptr, c_ptr + (data.bands_height * data.pixels_width));
    return thrust::reduce(c_vec.begin(), c_vec.end());
}

std::size_t PatchSystem::GetPatchNumbers(ImageSize size)
{
    return size.width * size.height;
}

CpuMatrix PatchSystem::GetPatchImage(std::size_t patch_idx)
{
    static constexpr std::size_t margin = PatchData::S / 2;

    const auto [size, img_data] = GetImageData(parent_img);

    const std::size_t band_offset = size.width * size.height;
    const std::size_t height_offset = size.width;

    CpuMatrix result{
        ImageSize{S, S, size.depth},
        std::make_shared<float[]>(S * S * size.depth)
    };

    const auto [center_x, center_y] = GeneratePatch(size, patch_idx);

    for (std::size_t band = 0; band < size.depth; band++)
    {
        for (int y = center_y - margin, iy=0; y < center_y + margin; ++y, ++iy)
        {
            for (int x = center_x - margin, ix=0; x < center_x + margin; ++x, ++ix)
            {
                float *value = result.data.get() + iy * S + ix + band * S * S;

                if (x < 0 || x >= size.width || y < 0 || y >= size.height)
                    *value = 0;
                else
                    *value = img_data[band * band_offset + y * height_offset + x];
            }
        }
    }

    return std::move(result);
}

PatchData PatchSystem::GeneratePatch(ImageSize size, std::size_t patch_idx)
{
    static constexpr std::size_t margin = PatchData::S / 2;

    std::size_t dy = patch_idx / (size.width);
    std::size_t dx = patch_idx % (size.width);

    return PatchData{dx, dy} ;
}
